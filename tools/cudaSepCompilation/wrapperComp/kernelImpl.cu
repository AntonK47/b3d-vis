#include "hip/hip_runtime.h"

#include "kernelHeader.cuh"

#include <cstdio>

#include "hip/hip_runtime.h"
#include ""

__global__ void waitKernel()
{
#if __CUDA_ARCH__ >= 700
	for (int i = 0; i < 1000; i++)
		__nanosleep(1000000U); // ls
#else
	const auto x = blockIdx.x * blockDim.x + threadIdx.x;
	const auto y = blockIdx.y * blockDim.y + threadIdx.y;
	if (add(x, y) == 0)
		printf(">>> __CUDA_ARCH__ must be 7.0 or higher!\n");
#endif
}

void waitKernelCall(dim3 gridDim, dim3 blockDim)
{
	waitKernel<<<gridDim, blockDim>>>();
}
