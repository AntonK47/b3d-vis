#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "deviceLib.cuh"

#include "cstdio"

__device__ __host__ auto add(const unsigned a, const unsigned b) -> unsigned
{
	return a + b;
}

__global__ void waitKernel()
{
	
	bar();
#if __CUDA_ARCH__ >= 700
	for (int i = 0; i < 1000; i++)
		__nanosleep(1000000U); // ls
#else
	const auto x = blockIdx.x * blockDim.x + threadIdx.x;
	const auto y = blockIdx.y * blockDim.y + threadIdx.y;
	if (add(x, y) == 0)
		printf(">>> __CUDA_ARCH__ must be 7.0 or higher!\n");
#endif
}

void waitKernelCall(dim3 gridDim, dim3 blockDim)
{
	waitKernel<<<gridDim, blockDim>>>();
}
