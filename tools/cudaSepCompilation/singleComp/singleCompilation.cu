
#include <cstdio>



#include "hip/hip_runtime.h"

__device__ __host__ auto add(const unsigned a, const unsigned b) -> unsigned
{
	return a + b;
}

__global__ void waitKernel()
{
#if __CUDA_ARCH__ >= 700
	for (int i = 0; i < 1000; i++)
		__nanosleep(1000000U); // ls
#else
	const auto x = blockIdx.x * blockDim.x + threadIdx.x;
	const auto y = blockIdx.y * blockDim.y + threadIdx.y;
	if (add(x,y) == 0)
		printf(">>> __CUDA_ARCH__ must be 7.0 or higher!\n");
#endif
}

auto main(const int argc, char** argv) -> int
{
	printf("1 + 1 = %d\n", add(1, 1));
	waitKernel<<<1, 1>>>();
}
