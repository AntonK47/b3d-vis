#include "hip/hip_runtime.h"
#include <>
#include <optix_device.h>
#include <owl/owl.h>

#include <nanovdb/util/HDDA.h>
#include <nanovdb/util/Ray.h>
#include "SharedStructs.h"
#include "nanovdb/NanoVDB.h"
#include "owl/common/math/vec.h"
#include "owl/owl_device.h"

using namespace b3d::renderer::nano;
using namespace owl;

extern "C" __constant__ LaunchParams optixLaunchParams;


struct PerRayData
{
	float t1;
	float result;
};

inline __device__ void confine(const nanovdb::BBox<nanovdb::Coord>& bbox, nanovdb::Vec3f& iVec)
{
	// NanoVDB's voxels and tiles are formed from half-open intervals, i.e.
	// voxel[0, 0, 0] spans the set [0, 1) x [0, 1) x [0, 1). To find a point's voxel,
	// its coordinates are simply truncated to integer. Ray-box intersections yield
	// pairs of points that, because of numerical errors, fall randomly on either side
	// of the voxel boundaries.
	// This confine method, given a point and a (integer-based/Coord-based) bounding
	// box, moves points outside the bbox into it. That means coordinates at lower
	// boundaries are snapped to the integer boundary, and in case of the point being
	// close to an upper boundary, it is move one EPS below that bound and into the volume.

	// get the tighter box around active values
	auto iMin = nanovdb::Vec3f(bbox.min());
	auto iMax = nanovdb::Vec3f(bbox.max()) + nanovdb::Vec3f(1.0f);

	// move the start and end points into the bbox
	float eps = 1e-7f;
	if (iVec[0] < iMin[0])
		iVec[0] = iMin[0];
	if (iVec[1] < iMin[1])
		iVec[1] = iMin[1];
	if (iVec[2] < iMin[2])
		iVec[2] = iMin[2];
	if (iVec[0] >= iMax[0])
		iVec[0] = iMax[0] - fmaxf(1.0f, fabsf(iVec[0])) * eps;
	if (iVec[1] >= iMax[1])
		iVec[1] = iMax[1] - fmaxf(1.0f, fabsf(iVec[1])) * eps;
	if (iVec[2] >= iMax[2])
		iVec[2] = iMax[2] - fmaxf(1.0f, fabsf(iVec[2])) * eps;
}

inline __hostdev__ void confine(const nanovdb::BBox<nanovdb::Coord>& bbox, nanovdb::Vec3f& iStart, nanovdb::Vec3f& iEnd)
{
	confine(bbox, iStart);
	confine(bbox, iEnd);
}


OPTIX_BOUNDS_PROGRAM(volumeBounds)
(const void* geometryData, owl::box3f& primitiveBounds, const int primitiveID)
{
	const auto& self = *static_cast<const GeometryData*>(geometryData);

	primitiveBounds = self.volume.worldAabb;
}

OPTIX_RAYGEN_PROGRAM(rayGeneration)()
{
	const auto& self = owl::getProgramData<RayGenerationData>();

	const int eyeIdx = optixLaunchParams.outputSurfaceIndex;
	const auto& camera = self.camera;
	const auto pixelId = owl::getLaunchIndex();

	const auto screen = (vec2f(pixelId) + vec2f(.5f)) / vec2f(self.frameBufferSize);//*2.0f -1.0f;

	owl::Ray ray;
	ray.origin = camera.position;
	ray.direction = normalize(camera.dir00 + screen.u * camera.dirDu + screen.v * camera.dirDv);

	PerRayData prd;
	owl::traceRay(self.world, ray, prd);

	vec3f color = { 0.2f, 0.1f, 0.0f };
	color *= prd.result;
	/*auto color = vec4f{};
	color.x = screen.x;
	color.y = screen.y;
	color.z = 0.0f;
	color.w = 1.0;*/
	surf2Dwrite(owl::make_rgba(color), self.frameBufferPtr[0] /* self.outputSurfaceArray[eyeIdx]*/,
				sizeof(uint32_t) * pixelId.x, pixelId.y);
}

OPTIX_MISS_PROGRAM(miss)()
{
	const auto pixelId = owl::getLaunchIndex();

	const auto& self = owl::getProgramData<MissProgramData>();

	auto& prd = owl::getPRD<vec3f>();
	const auto pattern = (pixelId.x / 8) ^ (pixelId.y / 8);
	prd = (pattern & 1) ? self.color1 : self.color0;
}

OPTIX_CLOSEST_HIT_PROGRAM(nano_closesthit)()
{
	const auto& geometry = owl::getProgramData<GeometryData>();
	const auto* grid = reinterpret_cast<const nanovdb::FloatGrid*>(geometry.volume.grid);

	const auto& tree = grid->tree();
	const auto& accessor = tree.getAccessor();

	auto& prd = owl::getPRD<PerRayData>();

	const auto rayOrigin = optixGetWorldRayOrigin();
	const auto rayDirection = optixGetWorldRayDirection();

	const auto t0 = optixGetRayTmax();
	const auto t1 = prd.t1;

	const auto rayWorld = nanovdb::Ray<float>(reinterpret_cast<const nanovdb::Vec3f&>(rayOrigin),
											  reinterpret_cast<const nanovdb::Vec3f&>(rayDirection));
	auto start = grid->worldToIndexF(rayWorld(t0));
	auto end = grid->worldToIndexF(rayWorld(t1));

	const auto bbox = grid->indexBBox();
	confine(bbox, start, end);


	const auto direction = end - start;
	const auto length = direction.length();
	const auto ray = nanovdb::Ray<float>(start, direction / length, 0.0f, length);
	auto ijk = nanovdb::RoundDown<nanovdb::Coord>(ray.start());


	auto hdda = nanovdb::HDDA<nanovdb::Ray<float>>(ray, accessor.getDim(ijk, ray));

	const auto opacity = 1.0f;
	auto transmittance = 1.0f;
	auto t = 0.0f;
	auto density = accessor.getValue(ijk) * opacity;
	while (hdda.step())
	{
		const auto dt = hdda.time() - t;
		transmittance *= expf(-density * dt);
		t = hdda.time();
		ijk = hdda.voxel();

		density = accessor.getValue(ijk) * opacity;
		hdda.update(ray, accessor.getDim(ijk, ray));
	}


	prd.result = transmittance;
}

OPTIX_INTERSECT_PROGRAM(nano_intersection)()
{
	const auto& geometry = owl::getProgramData<GeometryData>();
	const auto* grid = reinterpret_cast<const nanovdb::FloatGrid*>(geometry.volume.grid);

	const auto rayOrigin = optixGetObjectRayOrigin();
	const auto rayDirection = optixGetObjectRayDirection();

	const auto bbox = grid->indexBBox();
	auto t0 = optixGetRayTmin();
	auto t1 = optixGetRayTmax();
	const auto ray = nanovdb::Ray<float>(reinterpret_cast<const nanovdb::Vec3f&>(rayOrigin),
										 reinterpret_cast<const nanovdb::Vec3f&>(rayDirection), t0, t1);


	if (ray.intersects(bbox, t0, t1))
	{
		auto& prd = owl::getPRD<PerRayData>();
		prd.t1 = t1;
		optixReportIntersection(fmaxf(t0, optixGetRayTmin()), 0);
	}
}
