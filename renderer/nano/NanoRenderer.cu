#include "hip/hip_runtime.h"
#include <>
#include <optix_device.h>
#include <owl/owl.h>

#include <nanovdb/util/HDDA.h>
#include <nanovdb/util/Ray.h>
#include "SharedStructs.h"
#include "nanovdb/NanoVDB.h"
#include "owl/common/math/vec.h"

#include "owl/owl_device.h"

using namespace b3d::renderer::nano;
using namespace owl;

extern "C" __constant__ LaunchParams optixLaunchParams;


struct PerRayData
{
	vec3f color;
};

inline __device__ void confine(const nanovdb::BBox<nanovdb::Coord>& bbox, nanovdb::Vec3f& iVec)
{
	// NanoVDB's voxels and tiles are formed from half-open intervals, i.e.
	// voxel[0, 0, 0] spans the set [0, 1) x [0, 1) x [0, 1). To find a point's voxel,
	// its coordinates are simply truncated to integer. Ray-box intersections yield
	// pairs of points that, because of numerical errors, fall randomly on either side
	// of the voxel boundaries.
	// This confine method, given a point and a (integer-based/Coord-based) bounding
	// box, moves points outside the bbox into it. That means coordinates at lower
	// boundaries are snapped to the integer boundary, and in case of the point being
	// close to an upper boundary, it is move one EPS below that bound and into the volume.

	// get the tighter box around active values
	auto iMin = nanovdb::Vec3f(bbox.min());
	auto iMax = nanovdb::Vec3f(bbox.max()) + nanovdb::Vec3f(1.0f);

	// move the start and end points into the bbox
	float eps = 1e-7f;
	if (iVec[0] < iMin[0])
		iVec[0] = iMin[0];
	if (iVec[1] < iMin[1])
		iVec[1] = iMin[1];
	if (iVec[2] < iMin[2])
		iVec[2] = iMin[2];
	if (iVec[0] >= iMax[0])
		iVec[0] = iMax[0] - fmaxf(1.0f, fabsf(iVec[0])) * eps;
	if (iVec[1] >= iMax[1])
		iVec[1] = iMax[1] - fmaxf(1.0f, fabsf(iVec[1])) * eps;
	if (iVec[2] >= iMax[2])
		iVec[2] = iMax[2] - fmaxf(1.0f, fabsf(iVec[2])) * eps;
}

inline __hostdev__ void confine(const nanovdb::BBox<nanovdb::Coord>& bbox, nanovdb::Vec3f& iStart, nanovdb::Vec3f& iEnd)
{
	confine(bbox, iStart);
	confine(bbox, iEnd);
}


OPTIX_BOUNDS_PROGRAM(volumeBounds)
(const void* geometryData, owl::box3f& primitiveBounds, const int primitiveID)
{
	const auto& self = *static_cast<const GeometryData*>(geometryData);
	primitiveBounds = self.volume.worldAabb;
}

OPTIX_RAYGEN_PROGRAM(rayGeneration)()
{
	const auto& self = owl::getProgramData<RayGenerationData>();

	const int eyeIdx = optixLaunchParams.outputSurfaceIndex;
	const auto& camera = self.camera;
	const auto pixelId = owl::getLaunchIndex();

	const auto screen = (vec2f(pixelId) + vec2f(.5f)) / vec2f(self.frameBufferSize);//*2.0f -1.0f;

	owl::Ray ray;
	ray.origin = camera.position;
	ray.direction = normalize(camera.dir00 + screen.x * camera.dirDu + screen.y * camera.dirDv);

	PerRayData prd;
	owl::traceRay(self.world, ray, prd);

	const auto color = prd.color;
	surf2Dwrite(owl::make_rgba(color), self.frameBufferPtr[0],
				sizeof(uint32_t) * pixelId.x, pixelId.y);
}

OPTIX_MISS_PROGRAM(miss)()
{
	const auto pixelId = owl::getLaunchIndex();

	const auto& self = owl::getProgramData<MissProgramData>();

	auto& prd = owl::getPRD<PerRayData>();
	const auto pattern = (pixelId.x / 8) ^ (pixelId.y / 8);
	prd.color = (pattern & 1) ? self.color1 : self.color0;
}

OPTIX_CLOSEST_HIT_PROGRAM(nano_closesthit)()
{
	{
		auto& prd = owl::getPRD<PerRayData>();
		prd.color = vec3f(0.8,0.3,0.2);
		return;
	}
	
	const auto& geometry = owl::getProgramData<GeometryData>();
	const auto* grid = reinterpret_cast<const nanovdb::FloatGrid*>(geometry.volume.grid);

	const auto& tree = grid->tree();
	const auto& accessor = tree.getAccessor();


	const auto rayOrigin = optixGetWorldRayOrigin();
	const auto rayDirection = optixGetWorldRayDirection();

	const auto t0 = optixGetRayTmin();
	const auto t1 = optixGetRayTmax();

	const auto rayWorld = nanovdb::Ray<float>(reinterpret_cast<const nanovdb::Vec3f&>(rayOrigin),
											  reinterpret_cast<const nanovdb::Vec3f&>(rayDirection));
	auto start = grid->worldToIndexF(rayWorld(t0));
	auto end = grid->worldToIndexF(rayWorld(t1));

	const auto bbox = grid->indexBBox();
	confine(bbox, start, end);


	const auto direction = end - start;
	const auto length = direction.length();
	const auto ray = nanovdb::Ray<float>(start, direction / length, 0.0f, length);
	auto ijk = nanovdb::RoundDown<nanovdb::Coord>(ray.start());


	auto hdda = nanovdb::HDDA<nanovdb::Ray<float>>(ray, accessor.getDim(ijk, ray));

	const auto opacity = 1.0f;
	auto transmittance = 1.0f;
	auto t = 0.0f;
	auto density = accessor.getValue(ijk) * opacity;
	while (hdda.step())
	{
		const auto dt = hdda.time() - t;
		transmittance *= expf(-density * dt);
		t = hdda.time();
		ijk = hdda.voxel();

		density = accessor.getValue(ijk) * opacity;
		hdda.update(ray, accessor.getDim(ijk, ray));
	}

	auto& prd = owl::getPRD<PerRayData>();

	prd.color = vec3f(0.8,0.3,0.2) * transmittance;
}

OPTIX_INTERSECT_PROGRAM(nano_intersection)()
{
	const auto& geometry = owl::getProgramData<GeometryData>();
	const auto* grid = reinterpret_cast<const nanovdb::FloatGrid*>(geometry.volume.grid);

	const auto rayOrigin = optixGetObjectRayOrigin();
	const auto rayDirection = optixGetObjectRayDirection();

	const auto bbox = grid->indexBBox();
	auto t0 = optixGetRayTmin();
	auto t1 = optixGetRayTmax();
	const auto ray = nanovdb::Ray<float>(reinterpret_cast<const nanovdb::Vec3f&>(rayOrigin),
										 reinterpret_cast<const nanovdb::Vec3f&>(rayDirection), t0, t1);


	if (ray.intersects(bbox, t0, t1))
	{
		optixReportIntersection(fmaxf(t0, optixGetRayTmin()), 0);
	}
}
