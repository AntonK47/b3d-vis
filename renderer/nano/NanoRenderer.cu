#include "hip/hip_runtime.h"
#include <>
#include <optix_device.h>
#include <owl/owl.h>

#include <nanovdb/util/HDDA.h>
#include <nanovdb/util/Ray.h>
#include "SharedStructs.h"
#include "nanovdb/NanoVDB.h"
#include "owl/common/math/vec.h"

#include "owl/owl_device.h"

#include <array>

using namespace b3d::renderer::nano;
using namespace owl;

extern "C" __constant__ LaunchParams optixLaunchParams;

struct PerRayData
{
	vec3f color;
	float alpha;
};

inline __device__ void confine(const nanovdb::BBox<nanovdb::Coord>& bbox, nanovdb::Vec3f& iVec)
{
	// NanoVDB's voxels and tiles are formed from half-open intervals, i.e.
	// voxel[0, 0, 0] spans the set [0, 1) x [0, 1) x [0, 1). To find a point's voxel,
	// its coordinates are simply truncated to integer. Ray-box intersections yield
	// pairs of points that, because of numerical errors, fall randomly on either side
	// of the voxel boundaries.
	// This confine method, given a point and a (integer-based/Coord-based) bounding
	// box, moves points outside the bbox into it. That means coordinates at lower
	// boundaries are snapped to the integer boundary, and in case of the point being
	// close to an upper boundary, it is move one EPS below that bound and into the volume.

	// get the tighter box around active values
	auto iMin = nanovdb::Vec3f(bbox.min());
	auto iMax = nanovdb::Vec3f(bbox.max()) + nanovdb::Vec3f(1.0f);

	// move the start and end points into the bbox
	float eps = 1e-7f;
	if (iVec[0] < iMin[0])
		iVec[0] = iMin[0];
	if (iVec[1] < iMin[1])
		iVec[1] = iMin[1];
	if (iVec[2] < iMin[2])
		iVec[2] = iMin[2];
	if (iVec[0] >= iMax[0])
		iVec[0] = iMax[0] - fmaxf(1.0f, fabsf(iVec[0])) * eps;
	if (iVec[1] >= iMax[1])
		iVec[1] = iMax[1] - fmaxf(1.0f, fabsf(iVec[1])) * eps;
	if (iVec[2] >= iMax[2])
		iVec[2] = iMax[2] - fmaxf(1.0f, fabsf(iVec[2])) * eps;
}

inline __hostdev__ void confine(const nanovdb::BBox<nanovdb::Coord>& bbox, nanovdb::Vec3f& iStart, nanovdb::Vec3f& iEnd)
{
	confine(bbox, iStart);
	confine(bbox, iEnd);
}


OPTIX_BOUNDS_PROGRAM(volumeBounds)
(const void* geometryData, owl::box3f& primitiveBounds, const int primitiveID)
{
	const auto& self = *static_cast<const GeometryData*>(geometryData);
	primitiveBounds = self.volume.indexBox;
	/*self.volume.*/
}

OPTIX_RAYGEN_PROGRAM(hitCountRayGen)()
{
}

OPTIX_RAYGEN_PROGRAM(rayGeneration)()
{
	const auto& self = owl::getProgramData<RayGenerationData>();


	const auto& camera = optixLaunchParams.cameraData;
	const auto pixelId = owl::getLaunchIndex();

	const auto screen = (vec2f(pixelId) + vec2f(.5f)) / vec2f(self.frameBufferSize); //*2.0f -1.0f;

	owl::Ray ray;
	ray.origin = camera.pos;
	ray.direction = normalize(camera.dir_00 + screen.x * camera.dir_du + screen.y * camera.dir_dv);

	PerRayData prd;
	owl::traceRay(self.world, ray, prd);

	// const auto color = prd.color;
	const auto color = vec3f(0.8, 0.3, 0.2);
	const auto bg1 = vec3f(0.572f, 0.100f, 0.750f);
	const auto bg2 = vec3f(0.000f, 0.300f, 0.300f);
	const auto pattern = (pixelId.x / 8) ^ (pixelId.y / 8);
	const auto bgColor = (pattern & 1) ? bg1 : bg2;
	const auto a = prd.alpha;


	auto mix = (color * 1.0f - a) + a * bgColor;
	surf2Dwrite(owl::make_rgba(prd.color /*a * bgColor*/), optixLaunchParams.surfacePointer,
				sizeof(uint32_t) * pixelId.x, pixelId.y);
}

OPTIX_MISS_PROGRAM(miss)()
{
	const auto pixelId = owl::getLaunchIndex();

	const auto& self = owl::getProgramData<MissProgramData>();

	auto& prd = owl::getPRD<PerRayData>();
	const auto pattern = (pixelId.x / 8) ^ (pixelId.y / 8);
	prd.color = (pattern & 1) ? self.color1 : self.color0;
	prd.alpha = 1.0f;
}

OPTIX_CLOSEST_HIT_PROGRAM(nano_closestHit)()
{
	/*{
		auto& prd = owl::getPRD<PerRayData>();
		prd.color = vec3f(0.8,0.3,0.2);
		return;
	}*/
	

	const auto& geometry = owl::getProgramData<GeometryData>();
	auto* grid = reinterpret_cast<nanovdb::FloatGrid*>(geometry.volume.grid);

	const auto& accessor = grid->getAccessor();

	const auto rayOrigin = optixGetWorldRayOrigin();
	const auto rayDirection = optixGetWorldRayDirection();

	const auto t0 = optixGetRayTmax();
	const auto t1 = getPRD<float>();

	const auto rayWorld = nanovdb::Ray<float>(reinterpret_cast<const nanovdb::Vec3f&>(rayOrigin),
											  reinterpret_cast<const nanovdb::Vec3f&>(rayDirection));
	const auto rt0n = nanovdb::Vec3f{ rayWorld(t0) };
	const auto rt1n = nanovdb::Vec3f{ rayWorld(t1) };
	auto r0 = owl::vec3f{ rt0n[0], rt0n[1], rt0n[2] };
	auto r1 = owl::vec3f{ rt1n[0], rt1n[1], rt1n[2] };

	/*auto rt0 = xfmPoint(transform, r0);
	auto rt1 = xfmPoint(transform, r1);*/

	/*auto start = grid->worldToIndexF(nanovdb::Vec3f{ rt0.x, rt0.y, rt0.z});
	auto end = grid->worldToIndexF(nanovdb::Vec3f{ rt1.x, rt1.y, rt1.z});*/

	// auto map = grid->map();
	

	/*float matF[3][3] = { { transform.l.vx.x, transform.l.vy.x, transform.l.vz.x },
						 { transform.l.vx.y, transform.l.vy.y, transform.l.vz.y },
						 { transform.l.vx.z, transform.l.vy.z, transform.l.vz.z } };*/

	float transform[12];
	optixGetWorldToObjectTransformMatrix(transform);

	float invMatF[9]; /* = { inverseT.vx.x, inverseT.vy.x, inverseT.vz.x, inverseT.vx.y, inverseT.vy.y,
										  inverseT.vz.y, inverseT.vx.z, inverseT.vy.z, inverseT.vz.z };*/

	invMatF[0] = transform[0];
	invMatF[1] =  transform[1];
	invMatF[2] =  transform[2];
	invMatF[3] =  transform[4];
	invMatF[4] =  transform[5];
	invMatF[5] =  transform[6];
	invMatF[6] =  transform[8];
	invMatF[7] =  transform[9];
	invMatF[8] =  transform[10];

	float p[3];
	p[0] = transform[3];
	p[1] = transform[7];
	p[2] = transform[11];

	// map.set(matF, invMatF, p);

	// printf("============== %0.2f", grid->map().mInvMatF[4]);

	const auto startWorld = rayWorld(t0);
	const auto endWorld = rayWorld(t1);
	const auto a = nanovdb::Vec3f(startWorld[0] - p[0], startWorld[1] - p[1], startWorld[2] - p[2]);
	const auto b = nanovdb::Vec3f(endWorld[0] - p[0], endWorld[1] - p[1], endWorld[2] - p[2]);
	auto start = nanovdb::matMult(&invMatF[0], a);
	auto end = nanovdb::matMult(&invMatF[0], b);
	/*auto start = grid->worldToIndexF(rayWorld(t0));
	auto end = grid->worldToIndexF(rayWorld(t1));*/
	/*if (owl::getLaunchIndex().x == 0 && owl::getLaunchIndex().y == 0)
	{


		printf(": %0.2f \n", transform[0]);
		printf(": %0.2f \n", transform[1]);
		printf(": %0.2f \n", transform[2]);
		printf(": %0.2f \n", transform[3]);
		printf(": %0.2f \n", transform[4]);
		printf(": %0.2f \n", transform[5]);
		printf(": %0.2f \n", transform[6]);
		printf(": %0.2f \n", transform[7]);
		printf(": %0.2f \n", transform[8]);
		printf(": %0.2f \n", transform[9]);
		printf(": %0.2f \n", transform[10]);
		printf(": %0.2f \n", transform[11]);


		printf("=================\n");
	}*/


	/*auto start = map.applyInverseMapF(rayWorld(t0));
	auto end = map.applyInverseMapF(rayWorld(t1));*/

	// start[0] = rt0.x;
	// start[1] = rt0.y;
	// start[2] = rt0.z;

	// end[0] = rt1.x;
	// end[1] = rt1.y;
	// end[2] = rt1.z;

	const auto bbox = grid->indexBBox();
	confine(bbox, start, end);


	const auto direction = end - start;
	const auto length = direction.length();
	const auto ray = nanovdb::Ray<float>(start, direction / length, 0.0f, length);
	auto ijk = nanovdb::RoundDown<nanovdb::Coord>(ray.start());


	auto hdda = nanovdb::HDDA<nanovdb::Ray<float>>(ray, accessor.getDim(ijk, ray));

	const auto opacity = 1.0f; // 0.01f;//1.0.f;
	auto transmittance = 1.0f;
	auto t = 0.0f;
	auto density = accessor.getValue(ijk) * opacity;
	while (hdda.step())
	{
		const auto dt = hdda.time() - t;
		transmittance *= expf(-density * dt);
		t = hdda.time();
		ijk = hdda.voxel();
		const auto value = accessor.getValue(ijk);
		density = value * opacity;
		hdda.update(ray, accessor.getDim(ijk, ray));
	}

	auto& prd = owl::getPRD<PerRayData>();

	prd.color = vec3f(0.8, 0.3, 0.2) * transmittance;
	prd.alpha = transmittance;
}

OPTIX_INTERSECT_PROGRAM(nano_intersection)()
{
	const auto& geometry = owl::getProgramData<GeometryData>();
	const auto* grid = reinterpret_cast<const nanovdb::FloatGrid*>(geometry.volume.grid);

	const auto rayOrigin = optixGetObjectRayOrigin();
	const auto rayDirection = optixGetObjectRayDirection();

	const auto& bbox = grid->worldBBox();
	auto t0 = optixGetRayTmin();
	auto t1 = optixGetRayTmax();
	const auto ray = nanovdb::Ray<float>(reinterpret_cast<const nanovdb::Vec3f&>(rayOrigin),
										 reinterpret_cast<const nanovdb::Vec3f&>(rayDirection), t0, t1);


	if (ray.intersects(bbox, t0, t1))
	{
		auto& t = getPRD<float>();
		t = t1;

		optixReportIntersection(fmaxf(t0, optixGetRayTmin()), 0);
	}
}
