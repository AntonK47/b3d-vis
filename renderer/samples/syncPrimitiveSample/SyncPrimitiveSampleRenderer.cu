#include "hip/hip_runtime.h"
#include "SyncPrimitiveSampleRenderer.h"

#include "hip/hip_runtime.h"

#include ""
#include "cuda/std/cmath"

#include "math_functions.h"

__global__ auto writeVertexBuffer(hipSurfaceObject_t surface, unsigned int width, unsigned int height) -> void
{
	// gridDim: This variable is of type dim3 and contains the dimensions of the grid.
	// blockIdx: This variable is of type uint3 and contains the block index within the grid.
	// blockDim: This variable is of type dim3 and contains the dimensions of the block.
	// threadIdx: This variable is of type uint3 and contains the thread index within the block.

	const auto x = min(blockIdx.x * blockDim.x + threadIdx.x, width - 1);
	const auto y = min(blockIdx.y * blockDim.y + threadIdx.y, height - 1);

	auto val = uint32_t{ 0xFFFFFFFF };
	val = uint32_t{ 0xFF0000FF };
	if (x + y == 0)
	{
		printf("Hello from global thread 0");
	}
	surf2Dwrite(val, surface, x * sizeof(uint32_t), y);
}

__global__ void kernel()
{
#if __CUDA_ARCH__ >= 700
	for (int i = 0; i < 1000; i++)
		__nanosleep(1000000U); // ls
#else
	printf(">>> __CUDA_ARCH__ !\n");
#endif
}

void b3d::renderer::SyncPrimitiveSampleRenderer::onRender(const View& view)
{

	auto cudaRet = hipSuccess;

	auto waitParams = hipExternalSemaphoreWaitParams{};
	waitParams.flags = 0;
	waitParams.params.fence.value = view.fenceValue;
	cudaRet = hipWaitExternalSemaphoresAsync(&initializationInfo_.signalSemaphore, &waitParams, 1);

	// TODO: class members
	std::array<hipArray_t, 2> cudaArrays{};
	std::array<hipSurfaceObject_t, 2> cudaSurfaceObjects{};

	// Map and createSurface
	{
		cudaRet = hipGraphicsMapResources(1, const_cast<hipGraphicsResource_t*>(&view.colorRt.target));
		for (auto i = 0; i < view.colorRt.extent.depth; i++)
		{
			cudaRet = hipGraphicsSubResourceGetMappedArray(&cudaArrays[i], view.colorRt.target, i, 0);
			 
			hipResourceDesc resDesc{};
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = cudaArrays[i];
			cudaRet = hipCreateSurfaceObject(&cudaSurfaceObjects[i], &resDesc);
		}
	}

	// Execute Kernel
	{
		const auto gridDimXAdd = view.colorRt.extent.width % 2 == 0 ? 0 : 1;
		const auto gridDimYAdd = view.colorRt.extent.height % 2 == 0 ? 0 : 1;
		auto gridDim =
			dim3{ view.colorRt.extent.width / 32 + gridDimXAdd, view.colorRt.extent.height / 32 + gridDimYAdd };
		auto blockDim = dim3{ 32, 32 };
		writeVertexBuffer<<<gridDim, blockDim>>>(cudaSurfaceObjects[0], view.colorRt.extent.width, view.colorRt.extent.height);
		kernel<<<1, 1>>>();
		
		//cudaRet = hipGetLastError();
	}

	// test Copy - Uncomment to test
	if constexpr (false)
	{
		std::vector<uint32_t> hostMem;
		hostMem.resize(view.colorRt.extent.width * view.colorRt.extent.height);
		hipMemcpy2DFromArray(hostMem.data(), view.colorRt.extent.width * sizeof(uint32_t), cudaArrays[0], 0, 0,
							  view.colorRt.extent.width * sizeof(uint32_t), view.colorRt.extent.height,
							  hipMemcpyDeviceToHost);
	}

	// Destroy and unmap
	{
		for (auto i = 0; i < view.colorRt.extent.depth; i++)
		{
			cudaRet = hipDestroySurfaceObject(cudaSurfaceObjects[i]);
		}
		cudaRet = hipGraphicsUnmapResources(1, const_cast<hipGraphicsResource_t*>(&view.colorRt.target));
	}

	auto signalParams = hipExternalSemaphoreSignalParams{};
	signalParams.flags = 0;
	signalParams.params.fence.value = view.fenceValue;
	cudaRet = hipSignalExternalSemaphoresAsync(&initializationInfo_.waitSemaphore, &signalParams, 1);

	hipError_t rc = hipGetLastError();
	if (rc != hipSuccess)
	{
		fprintf(stderr, "error (%s: line %d): %d:  %s\n", __FILE__, __LINE__, rc, hipGetErrorString(rc));
		// OWL_RAISE("fatal cuda error");
	}
}

auto b3d::renderer::SyncPrimitiveSampleRenderer::onInitialize() -> void
{

}

auto b3d::renderer::SyncPrimitiveSampleRenderer::onDeinitialize() -> void
{
}

auto b3d::renderer::SyncPrimitiveSampleRenderer::onGui() -> void
{
}
