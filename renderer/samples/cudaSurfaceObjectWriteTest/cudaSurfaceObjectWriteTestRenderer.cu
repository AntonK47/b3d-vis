#include "hip/hip_runtime.h"
#include "CudaSurfaceObjectWriteTestRenderer.h"

#include "hip/hip_runtime.h"

#include ""
#include "cuda/std/cmath"

#include "math_functions.h"

using namespace b3d::renderer;

__global__ auto writeVertexBuffer(hipSurfaceObject_t surface, unsigned int width, unsigned int height) -> void
{
	// gridDim: This variable is of type dim3 and contains the dimensions of the grid.
	// blockIdx: This variable is of type uint3 and contains the block index within the grid.
	// blockDim: This variable is of type dim3 and contains the dimensions of the block.
	// threadIdx: This variable is of type uint3 and contains the thread index within the block.

	const auto x = min(blockIdx.x * blockDim.x + threadIdx.x, width - 1);
	const auto y = min(blockIdx.y * blockDim.y + threadIdx.y, height - 1);

	auto val = uint32_t{ 0xFFFFFFFF };
	
	if (x + y == 0)
	{
		// printf("Hello from global thread 0\n");
	}
	surf2Dwrite(val, surface, x * sizeof(uint32_t), y);
}


auto CudaSurfaceObjectWriteTestRenderer::onRender() -> void
{
	// TODO: class members
	std::array<hipArray_t, 2> cudaArrays{};
	std::array<hipSurfaceObject_t, 2> cudaSurfaceObjects{};

	const auto renderTargets = renderData_->get<RenderTargets>("renderTargets");

	auto cudaRet = hipSuccess;
	// Map and createSurface
	{
		cudaRet = hipGraphicsMapResources(1, const_cast<hipGraphicsResource_t*>(&renderTargets->colorRt.target));
		for (auto i = 0; i < renderTargets->colorRt.extent.depth; i++)
		{
			cudaRet = hipGraphicsSubResourceGetMappedArray(&cudaArrays[i], renderTargets->colorRt.target, i, 0);

			hipResourceDesc resDesc{};
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = cudaArrays[i];
			cudaRet = hipCreateSurfaceObject(&cudaSurfaceObjects[i], &resDesc);
		}
	}

	// Execute Kernel
	{
		const auto gridDimXAdd = renderTargets->colorRt.extent.width % 32 == 0 ? 0 : 1; 
		const auto gridDimYAdd = renderTargets->colorRt.extent.height % 32 == 0 ? 0 : 1; 
		auto gridDim = dim3{ renderTargets->colorRt.extent.width / 32 + gridDimXAdd,
							 renderTargets->colorRt.extent.height / 32 + gridDimYAdd };
		auto blockDim = dim3{ 32, 32 };
		writeVertexBuffer<<<gridDim, blockDim>>>(cudaSurfaceObjects[0], renderTargets->colorRt.extent.width,
												 renderTargets->colorRt.extent.height);
		hipDeviceSynchronize();
		cudaRet = hipGetLastError();
	}

	// test Copy - Uncomment to test
	if constexpr (false)
	{
		std::vector<uint32_t> hostMem;
		hostMem.resize(renderTargets->colorRt.extent.width * renderTargets->colorRt.extent.height);
		hipMemcpy2DFromArray(hostMem.data(), renderTargets->colorRt.extent.width * sizeof(uint32_t), cudaArrays[0], 0,
							  0, renderTargets->colorRt.extent.width * sizeof(uint32_t),
							  renderTargets->colorRt.extent.height,
							  hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		hipDeviceSynchronize();
	}

	// Destroy and unmap
	{
		for (auto i = 0; i < renderTargets->colorRt.extent.depth; i++)
		{
			cudaRet = hipDestroySurfaceObject(cudaSurfaceObjects[i]);
		}
		cudaRet = hipGraphicsUnmapResources(1, const_cast<hipGraphicsResource_t*>(&renderTargets->colorRt.target));
	}
}

auto CudaSurfaceObjectWriteTestRenderer::onInitialize() -> void
{
	
}

auto CudaSurfaceObjectWriteTestRenderer::onDeinitialize() -> void
{
	
}

auto CudaSurfaceObjectWriteTestRenderer::onGui() -> void
{
	
}
